#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include "enums.hpp"

__global__ void monteCarloKernel(
    float *results,
    int num_paths,
    float S, float K, float r, float sigma, float T,
    float barrier,
    int steps,
    OptionType option_type,
    BarrierType barrier_type,
    OptionStyle style,
    unsigned long long seed
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    float dt = T / steps;
    float st = S;
    float sum_for_average = 0.0f;
    bool knocked_out = false;

    for (int i = 0; i < steps; ++i) {
        float z = hiprand_normal(&state);
        st *= expf((r - 0.5f * sigma * sigma) * dt + sigma * sqrtf(dt) * z);

        if (option_type == OptionType::ASIAN) {
            sum_for_average += st;
        }

        if (option_type == OptionType::BARRIER) {
            if (barrier_type == BarrierType::UP_AND_OUT && st >= barrier) {
                knocked_out = true;
            }
            if (barrier_type == BarrierType::DOWN_AND_OUT && st <= barrier) {
                knocked_out = true;
            }
        }
    }

    float payoff = 0.0f;
    if (option_type == OptionType::EUROPEAN) {
        payoff = (style == OptionStyle::CALL) ? fmaxf(st - K, 0.0f) : fmaxf(K - st, 0.0f);
    }
    else if (option_type == OptionType::ASIAN) {
        float avg = sum_for_average / steps;
        payoff = (style == OptionStyle::CALL) ? fmaxf(avg - K, 0.0f) : fmaxf(K - avg, 0.0f);
    }
    else if (option_type == OptionType::BARRIER && !knocked_out) {
        payoff = (style == OptionStyle::CALL) ? fmaxf(st - K, 0.0f) : fmaxf(K - st, 0.0f);
    }

    results[idx] = expf(-r * T) * payoff;
}

void monteCarloKernelLauncher(
    at::Tensor results,
    int num_paths,
    float S, float K, float r, float sigma, float T,
    float barrier,
    int steps,
    OptionType option_type,
    BarrierType barrier_type,
    OptionStyle style,
    unsigned long long seed
)
{
    const int blockSize = 256;
    const int gridSize = (num_paths + blockSize - 1) / blockSize;

    monteCarloKernel<<<gridSize, blockSize>>>(
        results.data_ptr<float>(), num_paths,
        S, K, r, sigma, T, barrier, steps,
        option_type, barrier_type, style, seed
    );
}
